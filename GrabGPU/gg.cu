
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <string>
#include <algorithm>
#include <sstream>
#include <chrono>
#include <thread>

#define sleep(t) std::this_thread::sleep_for(std::chrono::milliseconds(t))

const float bytes_per_gb = (1 << 30);
const float ms_per_hour = 1000 * 3600;
const int max_grid_dim = (1 << 15);
const int max_block_dim = 1024;
const int max_sleep_time = 1e3;
const float sleep_interval = 1e16;
const int max_gpu_num = 32;

__global__ void default_script_kernel(char* array, size_t occupy_size) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= occupy_size) return;
  array[i]++;
}

void launch_default_script(char** array, size_t occupy_size,
                           std::vector<int>& grid_dim,
                           std::vector<int>& gpu_ids) {
  int gd = std::min(grid_dim[rand() % grid_dim.size()],
                    int(occupy_size / max_block_dim));
  for (int id : gpu_ids) {
    hipSetDevice(id);
    default_script_kernel<<<gd, max_block_dim, 0, NULL>>>(array[id],
                                                          occupy_size);
  }
}

void run_default_script(char** array, size_t occupy_size, float total_time,
                        std::vector<int>& gpu_ids) {
  printf("Running default script >>>>>>>>>>>>>>>>>>>>\n");
  srand(time(NULL));
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float time;
  size_t cnt = 0, sleep_time;
  std::vector<int> grid_dim;
  for (int i = 1; i <= max_grid_dim; i <<= 1) {
    grid_dim.push_back(i);
  }
  hipEventRecord(start, 0);
  while (true) {
    launch_default_script(array, occupy_size, grid_dim, gpu_ids);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    if (time / ms_per_hour > total_time) break;
    if (!((++cnt) % size_t(sleep_interval / occupy_size))) {
      cnt = 0;
      printf("Occupied time: %.2f hours\n", time / ms_per_hour);
      sleep_time = rand() % max_sleep_time + 1;
      sleep(sleep_time);
    }
  }
  hipEventDestroy(start);
  hipEventDestroy(stop);
  for (int id : gpu_ids) {
    hipFree(array[id]);
  }
}

void process_args(int argc, char** argv, size_t& occupy_size, float& total_time,
                  std::vector<int>& gpu_ids, std::string& script_path) {
  if (argc != 4 && argc != 5) {
    printf(
        "Arguments: <GPU Memory (GB)> <Occupied Time (h)> <GPU ID> <OPTIONAL: "
        "Script Path>\n");
    throw std::invalid_argument("Invalid argument number");
  }
  int gpu_num;
  hipGetDeviceCount(&gpu_num);
  int id;
  std::string s(argv[3]);
  std::replace(s.begin(), s.end(), ',', ' ');
  std::stringstream ss;
  ss << s;
  while (ss >> id) {
    gpu_ids.push_back(id);
  }
  if (gpu_ids.size() == 1 && gpu_ids[0] == -1) {
    gpu_ids[0] = 0;
    for (int i = 1; i < gpu_num; ++i) {
      gpu_ids.push_back(i);
    }
  }
  for (int i : gpu_ids) {
    if (i < 0 || i >= gpu_num) {
      printf("Invalid GPU ID (%d GPU in total): %d\n", i, gpu_num);
      throw std::invalid_argument("Invalid GPU ID");
    }
  }

  float occupy_mem;
  size_t total_size, avail_size;
  hipMemGetInfo(&avail_size, &total_size);
  sscanf(argv[1], "%f", &occupy_mem);
  sscanf(argv[2], "%f", &total_time);
  if (occupy_mem <= 0) {
    printf("GPU memory must be positive: %.2f\n", occupy_mem);
    throw std::invalid_argument("Invalid GPU memory");
  }
  if (total_time < 0) {
    printf("Occupied time must be positive: %.2f\n", total_time);
    throw std::invalid_argument("Invalid occupied time");
  }
  occupy_size = occupy_mem * bytes_per_gb;
  if (occupy_size > total_size) {
    printf("GPU memory exceeds maximum (%.2f GB): %.2f\n",
           total_size / bytes_per_gb, occupy_mem);
    throw std::invalid_argument("Exceed maximal GPU memory");
  }

  printf("GPU memory (GB): %.2f\n", occupy_mem);
  printf("Occupied time (h): %.2f\n", total_time);
  if (argc == 4) {
    printf("GPU ID: ");
    for (int id = 0; id < gpu_ids.size(); ++id) {
      printf("%d%c", gpu_ids[id], ",\n"[id == gpu_ids.size() - 1]);
    }
  } else {
    script_path = argv[4];
    printf("Script path: %s\n", script_path.c_str());
  }
}

void allocate_mem(char** array, size_t occupy_size, std::vector<int>& gpu_ids) {
  std::vector<bool> allocated(max_gpu_num, false);
  int cnt = 0;
  while (true) {
    printf("Try allocate GPU memory %d times >>>>>>>>>>>>>>>>>>>>\n", ++cnt);
    bool all_allocated = true;
    for (int id : gpu_ids) {
      if (!allocated[id]) {
        hipSetDevice(id);
        hipError_t status = hipMalloc(&array[id], occupy_size);
        size_t total_size, avail_size;
        hipMemGetInfo(&avail_size, &total_size);
        if (status != hipSuccess) {
          printf(
              "GPU-%d: Failed to allocate %.2f GB GPU memory (%.2f GB "
              "available)\n",
              id, occupy_size / bytes_per_gb, avail_size / bytes_per_gb);
          all_allocated = false;
        } else {
          allocated[id] = true;
          printf(
              "GPU-%d: Successfully allocate %.2f GB GPU memory (%.2f GB "
              "available)\n",
              id, occupy_size / bytes_per_gb, avail_size / bytes_per_gb);
        }
      }
    }
    if (all_allocated) break;
    sleep(5000);
  }
  printf("Successfully allocate memory on all GPUs!\n");
}

void run_custom_script(char** array, std::vector<int>& gpu_ids,
                       std::string script_path) {
  printf("Running custom script >>>>>>>>>>>>>>>>>>>>\n");
  for (int id : gpu_ids) {
    hipFree(array[id]);
  }
  std::string cmd = "sh " + script_path;
  std::system(cmd.c_str());
}

int main(int argc, char** argv) {
  size_t occupy_size;
  float total_time;
  std::vector<int> gpu_ids;
  std::string script_path;
  char* array[max_gpu_num];

  process_args(argc, argv, occupy_size, total_time, gpu_ids, script_path);
  allocate_mem(array, occupy_size, gpu_ids);

  if (argc == 4) {
    run_default_script(array, occupy_size, total_time, gpu_ids);
  } else {
    run_custom_script(array, gpu_ids, script_path);
  }

  return 0;
}
